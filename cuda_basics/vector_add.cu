
#include <hip/hip_runtime.h>
#include <cstddef>
#include <stdio.h>

namespace vecops {
	template<class T>
	__global__ 
	void adder(T* ar1, T* ar2, T* result, const std::size_t ar_size) {
		for (std::size_t index = 0; index < ar_size; index += 1) {
			//printf("Adding numbers: %d, %d\n", ar1[index], ar2[index]);
			result[index] = ar1[index] + ar2[index];
		}
	}
}